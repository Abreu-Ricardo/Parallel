/**
 * Trabalho 2
 *
 * Alunos: João Víctor Zárate, Julio Huang, Ricardo Abreu
 * 
 * Comando de compilação: 
 * nvcc dist_par.cu -o dist_par
 * Comando de execução :
 * ./dist_par entrada.txt
 *
 * O programa implementa apenas a solução inicial.
 * 
 * @file dist_par.cu
 * @author João Víctor Zárate, Julio Huang, Ricardo Abreu
 * @brief Trabalho 2 - Progrmação Paralela
 * @version 1.0
 * @date 2020-11-23
 * 
 * @copyright Copyright (c) 2020
 * 
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

char *aloca_sequencia(int n)
{
	char *seq;

	seq = (char *)malloc((n + 1) * sizeof(char));
	if (seq == NULL)
	{
		printf("\nErro na alocação de estruturas\n");
		exit(1);
	}
	return seq;
}

int *aloca_matriz(int n)
{
	int *seq;

	seq = (int *)malloc((n) * sizeof(int));
	if (seq == NULL)
	{
		printf("\nErro na alocação de estruturas\n");
		exit(1);
	}
	return seq;
}

void libera(int n, char *s, char *r, int *d)
{
	free(s);
	free(r);
	free(d);
}

//----------------------------------------------
__global__ void distancia(int *d, int n, int m, int i, char *s, char *r)
{

	int posi, t, a, b, c, min;
	int cima, diag, atras;
	int linha, coluna;
	__syncthreads();

	// Pegando cada cel para cada thread
	if (i >= n)
	{
		posi = (i * (m + 1)) - ((i - n) * (m));
		posi = posi - threadIdx.x * (m);
	}
	else
		posi = i * (m + 1) - threadIdx.x * (m) + m + 2;

	atras = posi - 1;
	cima = posi - (m + 1);
	diag = posi - (m + 2);

	printf("Para cada i=%d  m=%d\n", i, m);
	printf(" posi: %2d\n", posi);

	// Se é uma célula válida
	if ((d[posi] == 0 && posi > 0) && posi <= (n + 1) * (m + 1))
	{
		linha = (posi / (m + 1));
		coluna = posi - ((m + 1) * linha);
		//coluna = posi%(m+1); //((m+1)*linha);

		printf("Linha: %d coluna: %d posi: %d\n", linha, coluna, posi);
		// printf("%c<-- ashdashd\n", s[4]);

		t = (s[linha] != r[coluna] ? 1 : 0);
		a = d[atras] + 1;
		b = d[cima] + 1;
		c = d[diag] + t;

		// Calcula d[(i*(m+1)) + j] = min(a, b, c)
		if (a < b)
			min = a;
		else
			min = b;
		if (c < min)
			min = c;

		d[posi] = min;

		//printf("valores comp:%d %d %d  posi:%d no vetor %d\n", d[atras], d[cima], d[diag], d[posi], posi);
	}
}

int main(int argc, char **argv)
{
	int n,	// Tamanho da sequência s
		m,	// Tamanho da sequência r
		*d, // Matriz de distâncias com tamanho (n+1)*(m+1)
		i, j;
	char *s,		  // Sequência s de entrada (vetor com tamanho n+1)
		*r;			  // Sequência r de entrada (vetor com tamanho m+1)
	FILE *arqEntrada; // Arquivo texto de entrada

	if (argc != 2)
	{
		printf("O programa foi executado com argumentos incorretos.\n");
		printf("Uso: ./dist_seq <nome arquivo entrada>\n");
		exit(1);
	}

	// Abre arquivo de entrada
	arqEntrada = fopen(argv[1], "rt");

	if (arqEntrada == NULL)
	{
		printf("\nArquivo texto de entrada não encontrado\n");
		exit(1);
	}

	// Lê tamanho das sequências s e r
	fscanf(arqEntrada, "%d %d", &n, &m);

	// Aloca vetores s e r
	s = aloca_sequencia(n);
	r = aloca_sequencia(m);
	// Aloca matriz d
	d = aloca_matriz((n + 1) * (m + 1));

	// Lê sequências do arquivo de entrada
	s[0] = ' ';
	r[0] = ' ';
	fscanf(arqEntrada, "%s", &(s[1]));
	fscanf(arqEntrada, "%s", &(r[1]));
	// printf("STRING S->> %c", s[4]);

	// Fecha arquivo de entrada
	fclose(arqEntrada);

	struct timeval h_ini, h_fim;
	gettimeofday(&h_ini, 0);

	// Inicializa matriz de distâncias d
	for (i = 0; i <= m; i++)
	{
		d[i] = i;
	}

	for (j = 1; j <= n; j++)
	{
		d[(m * j) + j] = j;
	}

	// Calcula distância de edição entre sequências s e r, por anti-diagonais
	/*** Criando vars para a GPU ***/
	int *d_M;
	char *d_s, *d_r;

	hipMalloc((void **)&d_M, sizeof(int) * ((n + 1) * (m + 1)));
	hipMalloc((void **)&d_s, sizeof(char) * (n + 1));
	hipMalloc((void **)&d_r, sizeof(char) * (m + 1));

	hipMemcpy(d_M, d, sizeof(int) * ((n + 1) * (m + 1)), hipMemcpyHostToDevice);
	hipMemcpy(d_s, s, sizeof(char) * (n + 1), hipMemcpyHostToDevice);
	hipMemcpy(d_r, r, sizeof(char) * (m + 1), hipMemcpyHostToDevice);

	for (int i = 0; i < n + m + 1; i++)
	{
		distancia<<<1, n>>>(d_M, n, m, i, d_s, d_r);
	}

	hipDeviceSynchronize();

	hipMemcpy(d, d_M, sizeof(int) * ((n + 1) * (m + 1)), hipMemcpyDeviceToHost);

	gettimeofday(&h_fim, 0);
	// Tempo de execução na CPU em milissegundos
	long segundos = h_fim.tv_sec - h_ini.tv_sec;
	long microsegundos = h_fim.tv_usec - h_ini.tv_usec;
	double tempo = (segundos * 1e3) + (microsegundos * 1e-3);

	printf("Distância=%d\n", d[((n + 1) * (m + 1)) - 1]);
	printf("Tempo CPU = %.2fms\n", tempo);

	// Libera vetores s e r e matriz d
	libera(n, s, r, d);

	return 0;
}
